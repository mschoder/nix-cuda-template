#include <iostream>
#include <hip/hip_runtime.h>

// CUDA kernel function
__global__ void helloFromGPU()
{
  printf("Hello, World from the GPU!\n");
}

int main()
{
  std::cout << "Hello, World from the CPU!" << std::endl;

  // Launch the CUDA kernel with 1 block and 1 thread
  helloFromGPU<<<1, 1>>>();

  // Check for kernel launch errors
  hipError_t err = hipGetLastError();
  if (err != hipSuccess)
  {
    std::cerr << "CUDA Error: " << hipGetErrorString(err) << std::endl;
    return -1;
  }

  // Wait for GPU to finish before exiting
  err = hipDeviceSynchronize();
  if (err != hipSuccess)
  {
    std::cerr << "CUDA Sync Error: " << hipGetErrorString(err) << std::endl;
    return -1;
  }

  return 0;
}
